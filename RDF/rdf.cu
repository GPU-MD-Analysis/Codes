#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <cmath>
#include <string>
#include <cstdio>
#include <iomanip>
#include "dcdread.h"
#include "cudaerr.h"
#include<assert.h>
#include <boost/program_options.hpp>

namespace po = boost::program_options;
using namespace std;

__global__ void pair_gpu(const double* d_x, const double* d_y, const double* d_z, 
 unsigned long long int *d_g2, int numatm, int nconf, 
 const double xbox, const double ybox, const double zbox,
 int d_bin,  unsigned long long int bl);

int main(int argc , char* argv[] )
{
  double xbox,ybox,zbox;
  double* h_x,*h_y,*h_z;
  double* d_x,*d_y,*d_z;
  unsigned long long int *h_g2,*d_g2;
  int nbin;
  int nthreads,device;
  int numatm,nconf,inconf;
  unsigned long long int near2;
  string file;

  po::options_description desc("Options");
  desc.add_options()

  ("help,h","Display help")
  ("nconf,m",    po::value<int>(&inconf)->default_value(1000),     "number ofconfigurations")
  ("nbin,b",    po::value<int>(&nbin)->default_value(2000),     "number of bins")
  ("filename,f", po::value<string>(&file)->default_value("traj.dcd"),"trajectory file")
  ("device,d",   po::value<int>(&device)->default_value(1),   "CUDA device to use (0 or 1)")
  ("nthreads,k", po::value<int>(&nthreads)->default_value(128), "number of threads");

   po::variables_map vm;
   po::store(po::parse_command_line(argc, argv, desc), vm);
   if ( vm.count("help") || vm.count("h") || argc == 1) {
     std::cout << "RDF calculation Program"<<std::endl
     << desc << std::endl
     << "(defaults are in brackets)\n"
     << "Example: "
     <<argv[0]<<" -m 1000  -b 2000  -f traj.dcd  -d 1  -k 128 "
     <<std::endl;
     return 0;
   }
   po::notify(vm);
   cout<<"Using these options for "<<argv[0]<<endl
   <<" -m "<<inconf
   <<" -b "<<nbin
   <<" -f "<<file
   <<" -d "<<device
   <<" -k "<<nthreads
   <<endl;
   cout<<"\nHelp available using "<<argv[0]<<" -h\n\n";

HANDLE_ERROR (hipSetDevice(device));//pick the device to use
///////////////////////////////////////
  std::ifstream infile;
      infile.open(file.c_str());
      if(!infile){
        cout<<"file "<<file.c_str()<<" not found\n";
      return 1;
     }
  assert(infile);


  ofstream pairfile,stwo;
  pairfile.open("RDF.dat");
  stwo.open("Pair_entropy.dat");

/////////////////////////////////////////////////////////
  dcdreadhead(&numatm,&nconf,infile);
  cout<<"Dcd file has "<< numatm << " atoms and " << nconf << " frames"<<endl;
  if (inconf>nconf) cout << "nconf is reset to "<< nconf <<endl;
  else
  {nconf=inconf;}
  cout<<"Calculating RDF for " << nconf << " frames"<<endl;
////////////////////////////////////////////////////////

   unsigned long long int sizef= nconf*numatm*sizeof(double);
   unsigned long long int sizebin= nbin*sizeof(unsigned long long int);

  HANDLE_ERROR(hipHostAlloc((void **)&h_x, sizef, hipHostMallocDefault));
  HANDLE_ERROR(hipHostAlloc((void **)&h_y, sizef, hipHostMallocDefault));
  HANDLE_ERROR(hipHostAlloc((void **)&h_z, sizef, hipHostMallocDefault));
  HANDLE_ERROR(hipHostAlloc((void **)&h_g2, sizebin, hipHostMallocDefault));
  
  HANDLE_ERROR(hipMalloc((void**)&d_x, sizef));
  HANDLE_ERROR(hipMalloc((void**)&d_y, sizef));
  HANDLE_ERROR(hipMalloc((void**)&d_z, sizef));
  HANDLE_ERROR(hipMalloc((void**)&d_g2, sizebin));
  
  HANDLE_ERROR (hipPeekAtLastError());

  memset(h_g2,0,sizebin);
  HANDLE_ERROR(hipMemcpy(d_g2, h_g2, sizebin,hipMemcpyHostToDevice));
 
double ax[numatm],ay[numatm],az[numatm];
for (int i=0;i<nconf;i++) {
  dcdreadframe(ax,ay,az,infile,numatm,xbox,ybox,zbox);
  for (int j=0;j<numatm;j++){
    h_x[i*numatm+j]=ax[j];
    h_y[i*numatm+j]=ay[j];
    h_z[i*numatm+j]=az[j];
  }
}

    HANDLE_ERROR(hipMemcpy(d_x, h_x, sizef, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_y, h_y, sizef, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_z, h_z, sizef, hipMemcpyHostToDevice));

    cout<<"Reading of input file and transfer to gpu is completed"<<endl;

    near2=nthreads*(int(0.5*numatm*(numatm-1)/nthreads)+1);
    unsigned long long int nblock = (near2/nthreads);

    cout<<"Initial blocks are "<<nblock<<" "<<", now changing to ";
    
    int maxblock=65535;
     int bl;
     int blockloop= int(nblock/maxblock);
    if (blockloop != 0) {
        nblock=maxblock;
    }
    cout<<nblock<<" and will run over "<<(blockloop+1)<<" blockloops"<<endl;


   for (bl=0;bl<(blockloop+1);bl++) {
        cout <<bl<<endl;
   
        pair_gpu<<< nblock,nthreads >>>
       (d_x, d_y, d_z, d_g2, numatm, nconf, xbox, ybox, zbox, nbin, bl);

        HANDLE_ERROR (hipPeekAtLastError());
        HANDLE_ERROR(hipDeviceSynchronize());
    }
  
    HANDLE_ERROR(hipMemcpy(h_g2, d_g2, sizebin, hipMemcpyDeviceToHost));

double pi=acos(-1.0l);
double rho=(numatm)/(xbox*ybox*zbox);
double norm=(4.0l*pi*rho)/3.0l;
double rl,ru,nideal;
double g2[nbin];
double r,gr,lngr,lngrbond,s2=0.0l,s2bond=0.0l;
double box=min(xbox,ybox);
       box=min(box,zbox);
double del=box/(2.0l*nbin);
    for (int i=0;i<nbin;i++) {
      rl=(i)*del;
      ru=rl+del;
      nideal=norm*(ru*ru*ru-rl*rl*rl);
      g2[i]=(double)h_g2[i]/((double)nconf*(double)numatm*nideal);
      r=(i)*del;
      pairfile<<(i+0.5l)*del<<" "<<g2[i]<<endl;
      if (r<2.0l) {
         gr=0.0l;
        }
        else {
         gr=g2[i];
        }
      if (gr<1e-5) {
         lngr=0.0l;
        }
        else {
         lngr=log(gr);
        }
       
        if (g2[i]<1e-6) {
           lngrbond=0.0l;
          }
          else {
           lngrbond=log(g2[i]);
          }
         s2=s2-2.0l*pi*rho*((gr*lngr)-gr+1.0l)*del*r*r;
         s2bond=s2bond-2.0l*pi*rho*((g2[i]*lngrbond)-g2[i]+1.0l)*del*r*r;

    }
        stwo<<"s2 value is "<<s2<<endl;
        stwo<<"s2bond value is "<<s2bond<<endl;



  cout<<"\n\n\n#Freeing Device memory"<<endl;
  HANDLE_ERROR(hipFree(d_x));
  HANDLE_ERROR(hipFree(d_y));
  HANDLE_ERROR(hipFree(d_z));
  HANDLE_ERROR(hipFree(d_g2));

  cout<<"#Freeing Host memory"<<endl;
  HANDLE_ERROR(hipHostFree ( h_x ) );
  HANDLE_ERROR(hipHostFree ( h_y ) );
  HANDLE_ERROR(hipHostFree ( h_z ) );
  HANDLE_ERROR(hipHostFree ( h_g2 ) );

  cout<<"#Number of atoms processed: "<<numatm<<endl<<endl;
  cout<<"#Number of confs processed: "<<nconf<<endl<<endl;
  cout<<"#number of threads used: "<<nthreads<<endl<<endl;
  return 0;
}

__global__ void pair_gpu(
const double* d_x, const double* d_y, const double* d_z, 
unsigned long long int *d_g2, int numatm, int nconf, 
const double xbox,const double ybox,const double zbox,int d_bin,  unsigned long long int bl)
{
 double r,cut,dx,dy,dz;
 int ig2,id1,id2;
 double box;
 box=min(xbox,ybox);
 box=min(box,zbox);

 double del=box/(2.0*d_bin);
 cut=box*0.5;
 int thisi;
 double n;

 int i = blockIdx.x * blockDim.x + threadIdx.x;
 int maxi = min(int(0.5*numatm*(numatm-1)-(bl*65535*128)),(65535*128));

 if ( i < maxi ) {
 thisi=bl*65535*128+i;

 n=(0.5)*(1+ ((double) sqrt (1.0+4.0*2.0*thisi)));
 id1=int(n);
 id2=thisi-(0.5*id1*(id1-1));

 

 for (int frame=0;frame<nconf;frame++){
      dx=d_x[frame*numatm+id1]-d_x[frame*numatm+id2];
      dy=d_y[frame*numatm+id1]-d_y[frame*numatm+id2];
      dz=d_z[frame*numatm+id1]-d_z[frame*numatm+id2];

        dx=dx-xbox*(round(dx/xbox));
        dy=dy-ybox*(round(dy/ybox));
        dz=dz-zbox*(round(dz/zbox));
        
        r=sqrtf(dx*dx+dy*dy+dz*dz);
        if (r<cut) {
          ig2=(int)(r/del);
          atomicAdd(&d_g2[ig2],2) ;
        }
      }
    }
 }


